#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void calculoTrapezios(long X_intervalos, double X_inicial, double X_incremento, long Y_intervalos, double Y_inicial,
                                 double Y_incremento, double *resultados, long IntervaloPorThread){

    //Declarar uma variável compartilhada para soma total dentro de cada bloco
    __shared__ double soma_total_bloco[512];
    double x1, x2, soma_parcial = 0.0;

    if (threadIdx.x < 512) {
        soma_total_bloco[threadIdx.x] = 0.0;
    }

    //Intervalo de i para esta thread
    long iInicial = (threadIdx.x + blockIdx.x * 512) * IntervaloPorThread;
    long iFinal = min((X_intervalos*Y_intervalos), iInicial + IntervaloPorThread);

    //Sincronizar todos os threads antes de começar a computacao
    __syncthreads();

    for(long i=iInicial; i<iFinal; i++){
        soma_parcial = 0.0;
        long indice_x = i / Y_intervalos;
        long indice_y = i % Y_intervalos;

        x1 = X_inicial + (indice_x * X_incremento);
        x2 = X_inicial + (indice_x * X_incremento) + X_incremento;

        double Y1 = Y_inicial + (indice_y * Y_incremento);
        double Y2 = Y_inicial + (indice_y * Y_incremento) + Y_incremento;
        
        double g1 = sin(x1*x1 + Y1*Y1);
        double g2 = sin(x1*x1 + Y2*Y2);
        soma_parcial += (g1 + g2) * (Y2 - Y1)/2.0;

        g1 = sin(x2*x2 + Y1*Y1);
        g2 = sin(x2*x2 + Y2*Y2);
        soma_parcial += (g1 + g2) * (Y2 - Y1)/2.0;

     
        soma_total_bloco[threadIdx.x] += (soma_parcial * X_incremento)/2.0;
    }
    

    __syncthreads();

    // A thread 0 vai escrever o valor final na posição do vetor de resultados
    if (threadIdx.x == 0) {
        double soma_bloco = 0.0;
        for(long i=0; i<512; i++){
            soma_bloco += soma_total_bloco[i];
        }
        resultados[blockIdx.x] = soma_bloco;
    }
}

int main(){

    double soma_total = 0.0;
    double X_final = 1.5;
    double X_inicial = 0;
    long X_intervalos = 100000;
    double X_incremento = (X_final - X_inicial) / X_intervalos;

    double Y_final = 1.5;
    double Y_inicial = 0;
    long Y_intervalos = 100000;
    double Y_incremento = (Y_final - Y_inicial) / Y_intervalos;

    //Configuracao do kernel
    dim3 numBlocos(1000);//Quantidade de blocos
    dim3 threadsPorBloco(512);//Quantidade de threads por bloco

    double *resultados_host = (double *)malloc(numBlocos.x * sizeof(double));
    //Inicializando os resultados_host com 0.0
    memset(resultados_host, 0, numBlocos.x * sizeof(double));

    long IntervaloPorThread = (X_intervalos * Y_intervalos + (numBlocos.x * threadsPorBloco.x - 1)) / (numBlocos.x * threadsPorBloco.x);
    printf("Intervalo: %ld\n", IntervaloPorThread);

    //Alocar memoria para os resultados
    double *resultados;
    hipMalloc(&resultados, numBlocos.x * sizeof(double));
    hipMemcpy(resultados, resultados_host, numBlocos.x * sizeof(double), hipMemcpyHostToDevice);

    //Lancar o kernel
    calculoTrapezios<<<numBlocos, threadsPorBloco>>>(X_intervalos, X_inicial, X_incremento, Y_intervalos, Y_inicial,
                                                     Y_incremento, resultados, IntervaloPorThread);

    hipMemcpy(resultados_host, resultados, numBlocos.x * sizeof(double), hipMemcpyDeviceToHost);

    for (long i=0; i<numBlocos.x; i++) {
        soma_total += resultados_host[i];
    }

    printf("A soma total da integral é: %f\n", soma_total);
    //Liberar memoria
    hipFree(resultados);

    return 0;
}